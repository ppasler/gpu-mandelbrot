
#include <hip/hip_runtime.h>
__device__
unsigned int doIterations( double const realPart0,
                           double const imagPart0,
                           unsigned int const iters ) {
   // Initialize: z = z0
   double realPart = realPart0;
   double imagPart = imagPart0;
   unsigned int count = 0;
   // Loop until escape
   while ( ( count <= iters )
          && ((realPart*realPart + imagPart*imagPart) <= 4.0) ) {
      ++count;
      // Update: z = z*z + z0;
      double const oldRealPart = realPart;
      realPart = realPart*realPart - imagPart*imagPart + realPart0;
      imagPart = 2.0*oldRealPart*imagPart + imagPart0;
   }
   return count;
}
