#include "hip/hip_runtime.h"
/**
 * @file pctdemo_processMandelbrotElement.cu
 * 
 * CUDA code to calculate the Mandelbrot Set on a GPU.
 * 
 * Copyright 2011 The MathWorks, Inc.
 */

/** Work out which piece of the global array this thread should operate on */ 
__device__ size_t calculateGlobalIndex() {
    // Which block are we?
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    // Which thread are we within the block?
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    // How big is each block?
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    // Which thread are we overall?
    return localThreadIdx + globalBlockIndex*threadsPerBlock;

}

/** The actual Mandelbrot algorithm for a single location */ 
__device__ unsigned int doIterations( double const a, 
                                      double const b, 
                                      unsigned int const maxIters ) {
    // Initialize
    double x = a;
    double y = b;
    double x_old = x; 
    unsigned int count = 0;
    double x_old = x;
    // x_old --> save current x, since both formulas in the loop need  
    // to be calculated with same values

    // Loop until escape
    while ( ( count <= maxIters ) && ((x*x + y*y) <= 4.0) ) {
        ++count;
        
        x_old = x; 
        x = x*x - y*y + a;
        y = 2.0*x_old*y + b;
    }
    return count;
}


/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 */
__global__ void processMandelbrotElement( 
                      double * out, 
                      const double * x, 
                      const double * y,
                      const unsigned int maxIters, 
                      const unsigned int numel ) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();

    // If we're off the end, return now
    if (globalThreadIdx >= numel) {
        return;
    }
    
    // Get our X and Y coords
    double const a = x[globalThreadIdx];
    double const b = y[globalThreadIdx];

    // Run the itearations on this location
    unsigned int const count = doIterations( a, b, maxIters );
    out[globalThreadIdx] = log( double( count + 1 ) );
}
