
#include <hip/hip_runtime.h>
/**
 * @file pctdemo_processMandelbrotElement.cu
 * 
 * CUDA code to calculate the Mandelbrot Set on a GPU.
 * 
 * Copyright 2011 The MathWorks, Inc.
 */

/** Work out which piece of the global array this thread should operate on */ 
__device__ size_t calculateGlobalIndex() {
    // Which block are we?
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    // Which thread are we within the block?
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    // How big is each block?
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    // Which thread are we overall?
    return localThreadIdx + globalBlockIndex*threadsPerBlock;

}

/** The actual Mandelbrot algorithm for a single location */ 
__device__ unsigned int doIterations( double const x0, 
                                      double const y0, 
                                      double const a, 
                                      double const b,
                                      unsigned int const k,                                       
                                      unsigned int const maxIters ) {
    // Initialise: z = z0
    // depending on x0, y0 we calc the mandelbrot or julia set
    double x = x0;
    double y = y0;

    unsigned int count = 0;
    // Loop until escape
    while ( ( count <= maxIters )
            && ((x*x + y*y) <= 4.0) ) {
        ++count;
        // Update: z = z*z + z0;
        double const oldx = x;
        // real part
        x = x*x - y*y + a;
        // imaginary part
        y = 2.0*oldx*y + b;
    }
    return count;
}


/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 */
__global__ void processMandelbrotElementTest( 
                      double * out, 
                      const double * x, 
                      const double * y,
                      const double a, 
                      const double b,
                      const unsigned int k,
                      const unsigned int maxIters,
                      const unsigned int mandelbrot,
                      const unsigned int numel ) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();

    // If we're off the end, return now
    if (globalThreadIdx >= numel) {
        return;
    }
    
    // Get our X and Y coords
    double x0 = x[globalThreadIdx];
    double y0 = y[globalThreadIdx];

    double aVal = a;
    double bVal = b;

    if(mandelbrot == 1){
      aVal = a*x0;
      bVal = b*y0;
    }

    // Run the itearations on this location
    unsigned int const count = doIterations( x0, y0, aVal, bVal, k, maxIters );
    out[globalThreadIdx] = log( double( count + 1 ) );
}
